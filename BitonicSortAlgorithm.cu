#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include ""


__global__ void bitonicSort(int* arr, int i, int j, int n)
{

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int seqLength = (int)pow(2, (i - j + 1));
    int skip = (int)pow(2, (i - j));
    int temp;
 

    if (index % seqLength < skip)
    { 
        if ( index < n - 1)
        {
            
            // Ascending step
            if ((index / ((int)pow(2, i)) % 2) == 0)
            { 
                if (arr[index] > arr[index + skip])
                { 

                    temp = arr[index];
                    arr[index] = arr[index + skip];
                    arr[index + skip] = temp;
                }
            }
            else
            { 
                // Descending step
                if (arr[index] < arr[index + skip])
                {
                    temp = arr[index];
                    arr[index] = arr[index + skip];
                    arr[index + skip] = temp;  
                }
            }
        }
    }
}


int main(void)
{
    int n = 16;
    int size = n * sizeof(int);
    int threads = 16;
    int blocks = (n + threads - 1) / threads;
    int* arr = (int*)malloc(size);
    int* sorted_arr = (int*)malloc(size);
    int* d_arr;
 

    // Assign array elements and print it
    printf("Orignal array of %d elements :\n{", n);
    for (int i = 0; i < n; i++)
    {
      arr[i] = rand() % n;
      if(i == n-1)
        printf(" %d }", arr[i]);
        else
          printf(" %d ,", arr[i]);
    }
    
    // Allocate memory
    hipMalloc((void**)&d_arr, size);


    // Copy to device
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);


    // Call kernel (Bitonic sort) with (log(n)) / log(2)) times
    for (int step = 1; step <= (log(n)) / log(2); step++)
    {
        for (int stage = 1; stage <= step; stage++)
        {
            bitonicSort<<< blocks , threads >>>(d_arr, step, stage, n);

            hipMemcpy(sorted_arr, d_arr, size, hipMemcpyDeviceToHost); 

            /*printf("\nstep (%d) and stage (%d) : \n", step, stage);
            for (int k = 0; k < n; k++) {
                printf("[%d] ", sorted_arr[k]);
            }
            printf("\n");
            free(sorted_arr);
            sorted_arr = (int*)malloc(size); 
         */
        }
    }

    // Copy results back to host
    hipMemcpy(sorted_arr, d_arr, size, hipMemcpyDeviceToHost);

    // Print results
    printf("\nArray after being sorted :\n{");
    for (int i = 0; i < n-1; i++)
    {
          printf(" %d ,", sorted_arr[i]);
     }
    printf(" %d }",sorted_arr[n-1]);

    // Free variables
    hipFree(d_arr);
    free(arr);
    free(sorted_arr);

    return 0;
}